#include "hip/hip_runtime.h"
#include <iostream>
#include <hypergrid/raytracing.hpp>
#include <af/hip/hip_runtime.h>


namespace hypergrid
{

// CUDA Kernel: 
__global__
void raytracing_kernel(int* grid, int value,
                       size_t start_x, size_t start_y,
                       int* end_x, int* end_y, 
                       size_t dim_x, size_t dim_y,
                       size_t endpoints_size,
                       int connectivity = 8)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= endpoints_size) return;

    printf("blockIdx x: %i\t threadIdx x: %i\t idx: %i\n", blockIdx.x, threadIdx.x, idx);

    size_t bt_pix0 = 1;
    size_t bt_pix = bt_pix0;
    size_t istep = dim_x;

    int dx = end_x[idx] - (int)start_x;
    int dy = end_y[idx] - (int)start_y;
    int s = dx < 0 ? -1 : 0;

    dx = (dx ^ s) - s;
    bt_pix = (bt_pix ^ s) - s;

    size_t ptr_ = start_y * istep + start_x;

    s = dy < 0 ? -1 : 0;
    dy = (dy ^ s) - s;
    istep = (istep ^ s) - s;

    s = dy > dx ? -1 : 0;

    // conditional swaps
    dx ^= dy & s;
    dy ^= dx & s;
    dx ^= dy & s;

    bt_pix ^= istep & s;
    istep ^= bt_pix & s;
    bt_pix ^= istep & s;

    int minus_delta_ = 0;
    int plus_delta_ = 0;
    int minus_step_ = 0;
    int plus_step_ = 0;
    int error_ = 0;
    size_t size_ = 0;

    if (connectivity == 4)
    {
        error_ = 0;
        plus_delta_ = (dx + dx) + (dy + dy);
        minus_delta_ = -(dy + dy);
        plus_step_ = (int)(istep - bt_pix);
        minus_step_ = (int)bt_pix;
        size_ = dx + dy + 1;
    }
    else // connectivity == 8
    {
        error_ = dx - (dy + dy);
        plus_delta_ = dx + dx;
        minus_delta_ = -(dy + dy);
        plus_step_ = (int)istep;
        minus_step_ = (int)bt_pix;
        size_ = dx + 1;
    }
    size_t current_pos_y = (int)(ptr_ / dim_x);
    size_t current_pos_x = (int)((ptr_ - (current_pos_y * dim_x)));

    int mask = 0;

    printf("size_: %d\n", size_);
    for (int i = 0; i < size_; i++)
    {
        grid[ptr_] = value;

        mask = error_ < 0 ? -1 : 0;
        error_ += minus_delta_ + (plus_delta_ & mask);
        ptr_ += minus_step_ + (plus_step_ & mask);
        // printf("ptr_: %d", ptr_);
    }

}


void add_lines(af::array& grid, int value,
               size_t start_x, size_t start_y,
               af::array endpoints)
{
    // Ensure any JIT kernels have executed
    grid.eval();
    endpoints.eval();

    // Obtain device pointer from array object
    int *device_grid = grid.device<int>();
    int *device_end_x = endpoints(af::span, 0).device<int>();
    int *device_end_y = endpoints(af::span, 1).device<int>();

    // Determine ArrayFire's CUDA stream
    int af_id = af::getDevice();
    int cuda_id = afcu::getNativeId(af_id);
    hipStream_t af_cuda_stream = afcu::getStream(cuda_id);

    // Set arguments and run the kernel in ArrayFire's stream
    int block_size = 512;
    int grid_size = (endpoints.dims(0) + block_size - 1) / block_size;
    std::cout << "grid_size: " << grid_size << std::endl;
    std::cout << "block_size: " << block_size << std::endl;
    raytracing_kernel<<<grid_size, block_size, 0, af_cuda_stream>>>(device_grid, value,
                                                                    start_x, start_y,
                                                                    device_end_x, device_end_y,
                                                                    grid.dims(0), grid.dims(1),
                                                                    endpoints.dims(0),
                                                                    4);

    // Finish any pending CUDA operations
    hipDeviceSynchronize();

    // Return control of af::array memory to ArrayFire
    grid.unlock();
    grid.eval();
    std::cout << "LINESSSS" << std::endl;
    af_print(grid);
}


} // hypergrid namespace
