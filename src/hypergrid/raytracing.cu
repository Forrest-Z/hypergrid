#include "hip/hip_runtime.h"
#include <hypergrid/raytracing.hpp>
#include <af/hip/hip_runtime.h>


namespace hypergrid
{

__global__
void raytracing_kernel(int* grid, int value,
                       size_t start_x, size_t start_y,
                       int* end_x, int* end_y, 
                       size_t dim_x, size_t dim_y,
                       size_t endpoints_size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= endpoints_size ||
        end_x[idx] >= dim_x   ||
        end_y[idx] >= dim_x)
    {
        return;
    }

    size_t bt_pix0 = 1;
    size_t bt_pix = bt_pix0;
    size_t istep = dim_x;

    int dx = end_x[idx] - (int)start_x;
    int dy = end_y[idx] - (int)start_y;
    int s = dx < 0 ? -1 : 0;

    dx = (dx ^ s) - s;
    bt_pix = (bt_pix ^ s) - s;

    size_t ptr_ = start_y * istep + start_x;

    s = dy < 0 ? -1 : 0;
    dy = (dy ^ s) - s;
    istep = (istep ^ s) - s;

    s = dy > dx ? -1 : 0;

    // Conditional swaps
    dx ^= dy & s;
    dy ^= dx & s;
    dx ^= dy & s;

    bt_pix ^= istep & s;
    istep ^= bt_pix & s;
    bt_pix ^= istep & s;

    int minus_delta_ = 0;
    int plus_delta_ = 0;
    int minus_step_ = 0;
    int plus_step_ = 0;
    int error_ = 0;
    size_t size_ = 0;

    // Connectivity 4
    // error_ = 0;
    // plus_delta_ = (dx + dx) + (dy + dy);
    // minus_delta_ = -(dy + dy);
    // plus_step_ = (int)(istep - bt_pix);
    // minus_step_ = (int)bt_pix;
    // size_ = dx + dy + 1;

    // Connectivity 8
    error_ = dx - (dy + dy);
    plus_delta_ = dx + dx;
    minus_delta_ = -(dy + dy);
    plus_step_ = (int)istep;
    minus_step_ = (int)bt_pix;
    size_ = dx + 1;

    size_t grid_size = dim_x * dim_y;
    int mask = 0;
    for (int i = 0; i < size_; i++)
    {
        // Check if current cell is obstacle
        if (grid[ptr_] == 100) return;

        // Set cell value
        grid[ptr_] = value;

        // Update state
        mask = error_ < 0 ? -1 : 0;
        error_ += minus_delta_ + (plus_delta_ & mask);
        ptr_ += minus_step_ + (plus_step_ & mask);

        // Check if out of bounds
        if (ptr_ >= grid_size) return;
    }
}


void add_lines(af::array& grid, int value,
               size_t start_x, size_t start_y,
               af::array endpoints)
{
    // Ensure any JIT kernels have executed
    grid.eval();
    endpoints.eval();

    // Obtain device pointer from array object
    int *device_grid = grid.device<int>();
    int *device_end_x = endpoints(af::span, 0).device<int>();
    int *device_end_y = endpoints(af::span, 1).device<int>();

    // Determine ArrayFire's CUDA stream
    int af_id = af::getDevice();
    int cuda_id = afcu::getNativeId(af_id);
    hipStream_t af_cuda_stream = afcu::getStream(cuda_id);

    // Set arguments and run the kernel in ArrayFire's stream
    int block_size = 512;
    int grid_size = (endpoints.dims(0) + block_size - 1) / block_size;
    raytracing_kernel<<<grid_size, block_size, 0, af_cuda_stream>>>(device_grid, value,
                                                                    start_x, start_y,
                                                                    device_end_x, device_end_y,
                                                                    grid.dims(0), grid.dims(1),
                                                                    endpoints.dims(0));
    // Finish any pending CUDA operations
    hipDeviceSynchronize();

    // Return control of af::array memory to ArrayFire
    grid.unlock();
    grid.eval();
}

} // hypergrid namespace
